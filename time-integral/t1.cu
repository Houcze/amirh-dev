#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <fstream>
#include <string>
#include <cmath>
#include <io/netcdf>

__global__ void laplace_dev(double* phi, double* result, double* s1, double* s2, int2 d)
{
    int x_index = blockIdx.x * blockDim.x + threadIdx.x;
    //int y_index = blockIdx.y * blockDim.y + threadIdx.y;

    // int index = y_index * d.x + x_index;
    // int index = threadIdx.x;
    int index = x_index;
    int i;
    int j;

    if(index < d.x * d.y)
    {
        result[index] = 0.;
        s1[index] = 0.;
        s2[index] = 0.;

    }
    /*
    i = 1;
    j = 0;
    if((((index % d.y) + j) < d.y) && (((index % d.y) + j) >= 0) && (((index / d.y) + i) < d.x) && (((index / d.y) + i) >= 0) && (index < d.x * d.y))
    {

        s1[index + i * d.y + j] = phi[index];
    }
    */
    i = 1;
    j = 0;
    
    if(((index / d.y + i) < d.x) && ((index / d.y + i) >= 0) && ((index % d.y + j) < d.y) && ((index % d.y + j) >= 0))
    {
        s1[index + i * d.y + j] = phi[index];
    }

    
    /*
    i = -1;
    j = 0;
    if((((index % d.y) + j) < d.y) && (((index % d.y) + j) >= 0) && (((index / d.y) + i) < d.x) && (((index / d.y) + i) >= 0))
    {

        s2[index + i * d.y + j] = phi[index];
    }     
    */
    
    if(index < d.x * d.y)
    {
        result[index] = s1[index]; // + s2[index];
    }

    
    /*
    i = 0;
    j = 1;
    if((((index % d.y) + j) < d.y) && (((index % d.y) + j) >= 0) && (((index / d.y) + i) < d.x) && (((index / d.y) + i) >= 0))
    {

        result[index + i * d.y + j] += phi[index];
    }
    
    
    i = 0;
    j = -1;
    if((((index % d.y) + j) < d.y) && (((index % d.y) + j) >= 0) && (((index / d.y) + i) < d.x) && (((index / d.y) + i) >= 0))
    {

        result[index + i * d.y + j] += phi[index];
    }     
    */
    /*


  
    if(index < d.x * d.y)
    {
        result[index] -= 4 * phi[index];
    }
    */
    
}



/*
* 共享内存限制太多
*/
int laplace_host(double* phi, double* result, int N1, int N2)
{
    double* s1;
    double* s2;
    hipMalloc(&s1, N1 * N2 * sizeof(double));
    hipMalloc(&s2, N1 * N2 * sizeof(double));
    laplace_dev<<<std::ceil(double(N1 * N2) / 1024), 1024>>>(phi, result, s1, s2, make_int2(N1, N2));
    hipFree(s1);
    hipFree(s2);
    return EXIT_SUCCESS;
}

int main(void)
{
    int N1{10};
    int N2{10};
    double h{0.1};
    double* init_host;
    double* result_host;
    init_host = (double*) malloc(N1 * N2 * sizeof(double));   
    result_host = (double*) malloc(N1 * N2 * sizeof(double)); 
    double* init_dev;
    double* result_dev;
    hipMalloc(&init_dev, N1 * N2 * sizeof(double));
    hipMalloc(&result_dev, N1 * N2 * sizeof(double));

    for(int i=0; i<N1; i++)
    {
        for(int j=0; j<N2; j++)
        {
            init_host[i * N2 + j] = 1.;
            result_host[i * N2 + j] = 0.;
        }
    }

    for(int i=0; i<N1; i++)
    {
        for(int j=0; j<N2; j++)
        {
            std::cout << init_host[i * N2 + j] << '\t';
        }
        std::cout << '\n';
    }

    char filepath[] = "./input.nc";
    char varname[] = "temperature";
    // netcdf::ds(init_host, filepath, varname);
    /********************************************************************************************************/
    std::ofstream check_init;
    check_init.open("./init.txt");
    for(int j=0; j < N1; j++)
    {
        for(int k=0; k <N2; k++)
        {
            check_init << init_host[j * N2 + k] << '\t';
        }
        check_init << '\n';
    }
    check_init.close();
    /********************************************************************************************************/
    hipMemcpy(init_dev, init_host, sizeof(double) * N1 * N2, hipMemcpyHostToDevice);
    hipMemcpy(result_dev, result_host, sizeof(double) * N1 * N2, hipMemcpyHostToDevice);
    for(int i=0; i<1; i++)
    {
        std::cout << "Round " << i + 1 << std::endl;
        laplace_host(init_dev, result_dev, N1, N2);
        // hipMemcpy(init_dev, result_dev, sizeof(double) * N1 * N2, hipMemcpyDeviceToDevice);
        hipMemcpy(result_host, result_dev, sizeof(double) * N1 * N2, hipMemcpyDeviceToHost);
        std::ofstream outfile;
        outfile.open("./1r.txt");
        for(int j=0; j < N1; j++)
        {
            for(int k=0; k <N2; k++)
            {
                outfile << result_host[j * N2 + k] << '\t';
            }
            outfile << '\n';
        }
        outfile.close();

    }


}