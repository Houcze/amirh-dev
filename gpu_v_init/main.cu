#include <hip/hip_runtime.h>
#include <iostream>

__device__ double ad[5] = {1, 2, 3, 4, 5};
__device__ double *adh = ad;


int main(void)
{
	double *ah;
    ah = (double *) malloc(5 * sizeof(double));
	hipMemcpyFromSymbol(ah, HIP_SYMBOL(adh), sizeof(double) * 5);
    double *b;
    b = (double *) malloc(5 * sizeof(double));
    hipMemcpy(b, ah, 5 * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << b[0] << std::endl;
}