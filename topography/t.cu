#include <hip/hip_runtime.h>
#include <io/netcdf>
#include <iostream>
#include <Prop.h>

/*
    球面
    问题，我们在计算过程中不能出现任何意义的经纬度坐标
*/


int main(void)
{
    char path[] = {"./mola32.nc"};
    char varn[] = {"alt"};


    Prop::shape p;

    netcdf::ds_prop(&p, path, varn);
    
    int size = Prop::size(p);
    int dims = Prop::dims(p);

    std::cout << "Dims is " << dims << std::endl;
    std::cout << "Size is " << size << std::endl;

    double* topo;
    topo = (double *) malloc(size * sizeof(double));
    netcdf::ds(topo, path, varn);
    
    

}