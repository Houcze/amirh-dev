#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Func.h"
#define NodeSuccess 1

// Add
Func::Func(int m, int n, double (*f)(double, double))
{
    wid = m;
    len = n;
    f2 = f;
    InputNum = 2;
}

Func::Func(int m, int n, double (*f)(double))
{
    wid = m;
    len = n;
    f1 = f;
    InputNum = 1;
}

__global__ void Ops(double* x, double* result, F1 f1, int N1, int N2)
{
	int x_index = blockIdx.x * blockDim.x + threadIdx.x;
	int y_index = blockIdx.y * blockDim.y + threadIdx.y;	

	int index = x_index + y_index * N2;
	if(index < N1 * N2)
		result[index] = (*f1)(x[index]);
}


__global__ void Ops(double* x, double* y, double* result, F2 f2, int N1, int N2)
{
	int x_index = blockIdx.x * blockDim.x + threadIdx.x;
	int y_index = blockIdx.y * blockDim.y + threadIdx.y;	

	int index = x_index + y_index * N2;
	if(index < N1 * N2)
		result[index] = (*f2)(x[index], y[index]);
}


int Func::run()
{
    switch (InputNum)
    {
    case 1:
        Ops<<<ceil(wid * len / double(1024)), 1024>>>(x, result, *f1, wid, len);
        break;
    case 2:
        Ops<<<ceil(wid * len / double(1024)), 1024>>>(x, y, result, *f2, wid, len);
        break;    
    
    default:
        break;
    }
    return NodeSuccess;
}

int Func::Input(double* x1, double* x2)
{
    x = x1;
    y = x2;
    return EXIT_SUCCESS;
}

int Func::Input(double* x1)
{
    x = x1;
    return EXIT_SUCCESS;
}

int Func::rst(double* rst)
{
    result = rst;
    return EXIT_SUCCESS;
}