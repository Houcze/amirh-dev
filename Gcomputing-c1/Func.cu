#include <hip/hip_runtime.h>
#include "Func.h"
#define NodeSuccess 1

// Add
Func::Func(int m, int n, double (*f)(double, double))
{
    wid = m;
    len = n;
    f2 = f;
    InputNum = 2;
}

Func::Func(int m, int n, double (*f)(double))
{
    wid = m;
    len = n;
    f1 = f;
    InputNum = 1;
}

